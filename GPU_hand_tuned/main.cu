#include "hip/hip_runtime.h"
/*
Compiler options:
-O3 --std=c++14 --ptxas-options=-v --gpu-architecture=sm_35 -lineinfo -maxrregcount=128 -w --resource-usage
*/
#include <iostream>
#include <fstream>
#include <iomanip>
#include "DDEInit.h"
#include "GPUTimer.h"
#include "LorenzSystem.cuh"
#include "GPUDDESolver.cuh"

int main(int argc, char const *argv[])
{
		// get device information
		int dev = 0;
		hipDeviceProp_t deviceProp;
		CHECK(hipGetDeviceProperties(&deviceProp, dev));
		printf("Using Device %d: %s\n", dev, deviceProp.name);
		CHECK(hipSetDevice(dev));

		//constants
    const unsigned int nrOfInitialPoints = 50;
    const unsigned int nrOfSteps = 1000;
		const unsigned int nrOfPoints = nrOfSteps + 2 * nrOfInitialPoints;
		const unsigned int nrOfParameters = 65536;
		const unsigned int batchSize = nrOfParameters;
		const unsigned int nrOfBatches = (nrOfParameters + batchSize - 1)/batchSize;

		//memory sizes
		size_t tValsInitLen = nrOfInitialPoints;
		size_t xValsInitLen = nrOfInitialPoints * batchSize;
		size_t tValsLen = nrOfPoints;
		size_t xValsLen = nrOfPoints * batchSize;

		//fill integration settings struct
		integrationSettings intSettings;
		intSettings.nrOfInitialPoints = nrOfInitialPoints;
		intSettings.nrOfParameters = batchSize;
		intSettings.nrOfPoints = nrOfPoints;
		intSettings.nrOfSteps = nrOfSteps;
		intSettings.t0 = 13.0/28.0;
		intSettings.dt = intSettings.t0 / 400.0;

		//kernel configuration
		const unsigned int blocksize = 128;
		const unsigned int gridsize = (batchSize + blocksize - 1) / blocksize;
		dim3 block(blocksize);
		dim3 grid(gridsize);

		//parameter stuff, initial CPU and GPU memory
		double * parameterListHost = linspace(47, 50, nrOfParameters);
		double * parameterListDevice;
		hipMalloc((void**)&parameterListDevice,batchSize * sizeof(double));

		//discretize initial functions
		double * tInit = linspaceDisc(-13.0/28.0, 0.0, nrOfInitialPoints);
		double * y0Init = discretize(y0, tInit, nrOfInitialPoints);
		double * yd0Init = discretize(yd0, tInit, nrOfInitialPoints);

		//copy initial conditions to new bigger arrays
		double * y0 = new double[xValsInitLen];
		double * yd0 = new double[xValsInitLen];
		for (size_t i = 0; i < nrOfInitialPoints; i++)
		{
			for (size_t j = 0; j < batchSize; j++)
			{
				unsigned int idx = i*batchSize + j;
				y0[idx] = y0Init[i];
				yd0[idx] = yd0Init[i];
			}
		}

		//allocate GPU memory
		hipMalloc((void**)&intSettings.tVals, tValsLen*sizeof(double));
		hipMalloc((void**)&intSettings.yVals, xValsLen*sizeof(double));
		hipMalloc((void**)&intSettings.ydVals, xValsLen*sizeof(double));

		//copy the initial values to gpu memory
		hipMemcpy(intSettings.tVals, tInit,tValsInitLen*sizeof(double),hipMemcpyHostToDevice);
		hipMemcpy(intSettings.yVals, y0,xValsInitLen*sizeof(double),hipMemcpyHostToDevice);
		hipMemcpy(intSettings.ydVals, yd0,xValsInitLen*sizeof(double),hipMemcpyHostToDevice);

		//information about the run
		printf("Memory size: %zd MB \n",(xValsLen*4+tValsLen)*sizeof(double)/1024/1024);
		printf("Launching kernel with <<<%d,%d>>> in %d batches\n",gridsize,blocksize,nrOfBatches);

		//save to file
		std::ofstream ofs("GPU_endvalues_3.txt");
		int id = nrOfInitialPoints + nrOfSteps;

		//execution in batches
		double tStart = seconds();

		//execute in batches
		for (size_t k = 0; k < nrOfBatches; k++)
		{
			CHECK(hipMemcpy(parameterListDevice,parameterListHost + k*batchSize,batchSize * sizeof(double),hipMemcpyHostToDevice));

			//launch kernel
			solver<<<grid,block>>>(intSettings, parameterListDevice);
			CHECK(hipDeviceSynchronize());

			//copy back to global memory
			double * yRef = new double[xValsLen];
			CHECK(hipMemcpy(yRef,intSettings.yVals,xValsLen*sizeof(double),hipMemcpyDeviceToHost));

			for (size_t i = 0; i < batchSize; i++)
			{
					double y = yRef[id*batchSize + i];
					ofs << parameterListHost[k*batchSize + i] <<"\t" << y << "\n";
			}

			delete yRef;
		}
		double tEnd = seconds();
		printf("Execution finished for p = %d parameters in t = %lf s \n", nrOfParameters, (tEnd - tStart) );
		ofs.flush();
		ofs.close();

		//free gpu memomory
		hipFree(parameterListDevice);

		//delete cpu memory
		delete parameterListHost;
		delete tInit, y0Init, yd0Init;
}
