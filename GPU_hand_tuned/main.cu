#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <iomanip>
#include "DDEInit.h"
#include "GPUTimer.h"
#include "LorenzSystem.cuh"
#include "GPUDDESolver.cuh"

int main(int argc, char const *argv[])
{
		// get device information
		int dev = 0;
		hipDeviceProp_t deviceProp;
		CHECK(hipGetDeviceProperties(&deviceProp, dev));
		printf("Using Device %d: %s\n", dev, deviceProp.name);
		CHECK(hipSetDevice(dev));

		//constants
    const unsigned int nrOfInitialPoints = 100;
    const unsigned int nrOfSteps = 10000;
		const unsigned int nrOfPoints = nrOfSteps + 2 * nrOfInitialPoints;
		const unsigned int nrOfParameters = 32768;
		const unsigned int batchSize = 4096;
		const unsigned int nrOfBatches = (nrOfParameters + batchSize - 1)/batchSize;

		//memory sizes
		size_t tValsInitLen = nrOfInitialPoints;
		size_t xValsInitLen = nrOfInitialPoints * batchSize;
		size_t tValsLen = nrOfPoints;
		size_t xValsLen = nrOfPoints * batchSize;

		//fill integration settings struct
		integrationSettings intSettings;
		intSettings.nrOfInitialPoints = nrOfInitialPoints;
		intSettings.nrOfParameters = batchSize;
		intSettings.nrOfPoints = nrOfPoints;
		intSettings.nrOfSteps = nrOfSteps;
		intSettings.t0 = 13.0/28.0;

		//kernel configuration
		const unsigned int blocksize = 64;
		const unsigned int gridsize = (batchSize + blocksize - 1) / blocksize;
		dim3 block(blocksize);
		dim3 grid(gridsize);

		//parameter stuff, initial CPU and GPU memory
		double * parameterListHost = linspace(47, 50, nrOfParameters);
		double * parameterListDevice;
		hipMalloc((void**)&parameterListDevice,batchSize * sizeof(double));

		//discretize initial functions
		double * tInit = linspaceDisc(-1.0, 0.0, nrOfInitialPoints);
		double * x0Init = discretize(x0, tInit, nrOfInitialPoints);
		double * y0Init = discretize(y0, tInit, nrOfInitialPoints);
		double * yd0Init = discretize(yd0, tInit, nrOfInitialPoints);
		double * z0Init = discretize(z0, tInit, nrOfInitialPoints);

		//copy initial conditions to new bigger arrays
		double * x0 = new double[xValsInitLen];
		double * y0 = new double[xValsInitLen];
		double * yd0 = new double[xValsInitLen];
		double * z0 = new double[xValsInitLen];
		for (size_t i = 0; i < nrOfInitialPoints; i++)
		{
			for (size_t j = 0; j < batchSize; j++)
			{
				unsigned int idx = i*batchSize + j;
				x0[idx] = x0Init[i];
				y0[idx] = y0Init[i];
				yd0[idx] = yd0Init[i];
				z0[idx] = z0Init[i];
			}
		}

		//allocate GPU memory
		hipMalloc((void**)&intSettings.tVals, tValsLen*sizeof(double));
		hipMalloc((void**)&intSettings.xVals, xValsLen*sizeof(double));
		hipMalloc((void**)&intSettings.yVals, xValsLen*sizeof(double));
		hipMalloc((void**)&intSettings.ydVals, xValsLen*sizeof(double));
		hipMalloc((void**)&intSettings.zVals, xValsLen*sizeof(double));

		//copy the initial values to gpu memory
		hipMemcpy(intSettings.tVals, tInit,tValsInitLen*sizeof(double),hipMemcpyHostToDevice);
		hipMemcpy(intSettings.xVals, x0,xValsInitLen*sizeof(double),hipMemcpyHostToDevice);
		hipMemcpy(intSettings.yVals, y0,xValsInitLen*sizeof(double),hipMemcpyHostToDevice);
		hipMemcpy(intSettings.ydVals, yd0,xValsInitLen*sizeof(double),hipMemcpyHostToDevice);
		hipMemcpy(intSettings.zVals, z0,xValsInitLen*sizeof(double),hipMemcpyHostToDevice);

		//information about the run
		printf("Memory size: %zd MB \n",(xValsLen*4+tValsLen)*sizeof(double)/1024/1024);
		printf("Launching kernel with <<<%d,%d>>> in %d batches\n",gridsize,blocksize,nrOfBatches);

		printf("Mesh: \t");
		for (size_t i = 0; i < 3; i++)
		{
			intSettings.mesh[i] = (i+1)*intSettings.t0;
			printf("%8.5lf\t",intSettings.mesh[i]);
		}
		printf("\n");

		//save to file
		std::ofstream ofs("GPU_endvalues_3.txt");
		int id = nrOfInitialPoints + 2 + nrOfSteps;

		//execution in batches
		double tStart = seconds();

		//execute in batches
		for (size_t k = 0; k < nrOfBatches; k++)
		{
			CHECK(hipMemcpy(parameterListDevice,parameterListHost + k*batchSize,batchSize * sizeof(double),hipMemcpyHostToDevice));

			//launch kernel
			solver<<<grid,block>>>(intSettings, parameterListDevice);
			CHECK(hipDeviceSynchronize());

			//copy back to global memory
			double * xRef = new double[xValsLen];
			hipMemcpy(xRef,intSettings.xVals,xValsLen*sizeof(double),hipMemcpyDeviceToHost);

			for (size_t i = 0; i < batchSize; i++)
			{
					double x = xRef[id*batchSize + i];
					ofs << parameterListHost[k*batchSize + i] <<"\t" << x << "\n";
			}

			delete xRef;
		}
		double tEnd = seconds();
		printf("Execution finished for p = %d parameters in t = %lf s \n", nrOfParameters, (tEnd - tStart) );
		ofs.flush();
		ofs.close();

		//free gpu memomory
		hipFree(parameterListDevice);

		//delete cpu memory
		delete parameterListHost;
		delete tInit, x0Init, y0Init, yd0Init, z0Init;
}
